#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cstring> 

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "../stb/stb_image.h"
#include "../stb/stb_image_write.h"

#define DICESIZE 10

#define CUDA_CALL(x, message) {if((x) != hipSuccess) { \
    printf("Error - %s(%d)[%s]: %s\n", __FILE__, __LINE__, message, hipGetErrorString(x)); \
    exit(EXIT_FAILURE); }}

#define DICE(dimg, idx, cond) \
    (dimg[idx] = dimg[idx + 1] = dimg[idx + 2] = (cond) ? 0 : 255)

__global__ void img2d6Kernel(unsigned char *dimg, int width, int height, int channels){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ unsigned int avg[DICESIZE - 1][DICESIZE - 1];

    if(row >= height || col >= width)
        return;

    int idx = (row * width + col) * channels;
    if(threadIdx.x == DICESIZE - 1 || threadIdx.y == DICESIZE - 1){
        dimg[idx] = dimg[idx + 1] = dimg[idx + 2] = 0;
        return;
    }

    unsigned char grey = (unsigned char)(0.299f * dimg[idx] + 0.587f * dimg[idx + 1] + 0.114f * dimg[idx + 2]);
    int x = threadIdx.x, y = threadIdx.y;
    avg[x][y] = grey;
    __syncthreads();
    if(x == 0){
        for(int i = 1; i < DICESIZE - 1; i++){
            avg[0][y] += avg[i][y];
        }   
    }
    __syncthreads();
    if(x == 0 && y == 0){
        for(int i = 1; i < DICESIZE - 1; i++){
            avg[0][0] += avg[0][i];
        }
        int d = avg[0][0] / ((DICESIZE - 1) * (DICESIZE - 1)) *6 / 255;
        avg[0][0] = d > 5 ? 6 : d + 1;;
    }
    __syncthreads();

    int d = 7 - avg[0][0];
    switch(d){
        case 1:
            DICE(dimg, idx, x >= 3 && x <= 5 && y >= 3 & y <= 5);
            break;
        case 2:
            DICE(dimg, idx, (x >= 6 && y < 3) || (x < 3 && y >= 6));
            break;
        case 3:
            DICE(dimg, idx, (x >= 3 && x <= 5 && y >= 3 & y <= 5) || (x >= 6 && y < 3) || (x < 3 && y >= 6));
            break;
        case 4:
            DICE(dimg, idx, (x >= 6 && y < 3) || (x < 3 && y >= 6) || (x < 3 && y < 3) || (x >= 6 && y >= 6));
            break;
        case 5:
            DICE(dimg, idx, (x >= 6 && y < 3) || (x < 3 && y >= 6) || (x < 3 && y < 3) || (x >= 6 && y >= 6) || (x >=3 && x <= 5 && y >= 3 & y <= 5));
            break;
        case 6:
            DICE(dimg, idx, y < 3 || y >= 6);
            break;
    }
}


int main(int argc, char** argv) {

    if (argc != 2) {
        printf("Usage: %s <image_path>\n", argv[0]);
        return -1;
    }
    const char* input_img = argv[1];
    int len = strlen(input_img);
    char* output_img = new char[len + 7];
    strcpy(output_img, "output_");
    strcat(output_img, input_img);

    int width, height, channels;
    unsigned char *himg = stbi_load(input_img, &width, &height, &channels, 3);
    unsigned char *dimg;
    int size = width * height * channels * sizeof(unsigned char);

    CUDA_CALL(hipMalloc((void **)&dimg, size), "hipMalloc - dimg");

    CUDA_CALL(hipMemcpy(dimg, himg, size, hipMemcpyHostToDevice), "hipMemcpy - himg -> dimg");

    int nn = DICESIZE;
    dim3 block_size(nn, nn);
    dim3 grid_size(
        (height - 1) / block_size.x + 1,
        (width - 1) / block_size.y + 1
    );

    img2d6Kernel<<<grid_size, block_size>>>(dimg, width, height, channels);

    CUDA_CALL(hipMemcpy(himg, dimg, size, hipMemcpyDeviceToHost), "hipMemcpy - dimg -> himg");

    stbi_write_png(output_img, width, height, channels, himg, width * channels);

    CUDA_CALL(hipFree(dimg), "hipFree - dimg")
    printf("Saved to %s\n", output_img);

    delete output_img;

    return 0;
}
